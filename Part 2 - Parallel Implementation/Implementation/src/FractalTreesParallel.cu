//#include "shared/jbutil.h"

#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cmath>
#include <map>
#include <climits>
#include <assert.h>

using std::cout;
using std::cerr;
using std::endl;
using std::string;

//using jbutil::matrix;
//using jbutil::image;

//__global__ void populate_sin_cos_maps(*sin_map, *cos_map);
__global__ void populate_sin_cos_maps(float *sin_map, float *cos_map){
		float rad = threadIdx.x * 0.01745329251f; // small number is pi/180
	 sincosf(rad, &sin_map[threadIdx.x], &cos_map[threadIdx.x]);
}

__global__ void calculate_points(
		short *angles,
		float *pointsX,
		float *pointsY,
		const int iterations,
		float line_length,
		const float length_multiplier,
		const int rotation_angle_degrees
)
{
		// initialize the cos and sin maps (Note, blockdim must be 512
		__shared__ float sin_map[512];
		__shared__ float cos_map[512];
		float rad = threadIdx.x * 0.01745329251f; // small number is pi/180
		sincosf(rad, &sin_map[threadIdx.x], &cos_map[threadIdx.x]);
		__syncthreads();

		// initialize the angles and points list
		const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
		printf("%d\n",index);
		for(int i = 0; i < iterations; ++i){
				if( index < (1 << i) ){ // first we compute first 2, then next 4, then next 8, etc
						unsigned int array_index = index*2 + (2<<i); // first 2 already initialized
						unsigned int array_index_plus_1 = array_index + 1; // first 2 already initialized
						unsigned int array_index_div_2 = array_index >> 1;

						angles[array_index] = ( (angles[array_index_div_2] + rotation_angle_degrees) ) % 360;
						angles[array_index_plus_1] = ( (angles[array_index_div_2 % 360] - rotation_angle_degrees) + 360 ) % 360;

						pointsX[array_index] = pointsX[array_index_div_2] + line_length * sin_map[ (angles[array_index]) ];
					 pointsY[array_index] = pointsY[array_index_div_2] + line_length * cos_map[ (angles[array_index]) ];
						pointsX[array_index_plus_1] = pointsX[array_index_div_2] + line_length * sin_map[ (angles[array_index_plus_1]) ];
						pointsY[array_index_plus_1] = pointsY[array_index_div_2] + line_length * cos_map[ (angles[array_index_plus_1]) ];
				}
				__syncthreads();
				line_length *= length_multiplier;
				__syncthreads();
		}
}

__global__ void calculateMin(float *points, float *storeList, float *retValue, const int iterations, unsigned long no_of_threads){
		const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
		if(index < no_of_threads){
			 storeList[index] = points[ index*2 + (points[index*2] > points[index*2 + 1]) ];
		}
		
		no_of_threads /= 2;
		__syncthreads();

		for(int i = 1; i < iterations; ++i){ // start i from 1 as the first iteration has already been done
				if(index < no_of_threads){
						storeList[index] = storeList[ index*2 + (storeList[index*2] > storeList[index*2 + 1]) ];
				}
				no_of_threads/=2;
				__syncthreads();
		}
		
		*retValue = storeList[0];
}

__global__ void calculateMax(float *points, float *storeList, float *retValue, const int iterations, unsigned long no_of_threads){
		const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
		if(index < no_of_threads){
			 storeList[index] = points[ index*2 + (points[index*2] < points[index*2 + 1]) ];
		}
		
		no_of_threads /= 2;
		__syncthreads();

		for(int i = 1; i < iterations; ++i){ // start i from 1 as the first iteration has already been done
				if(index < no_of_threads){
						storeList[index] = storeList[ index*2 + (storeList[index*2] < storeList[index*2 + 1]) ];
				}
				no_of_threads/=2;
				__syncthreads();
		}
		
		*retValue = storeList[0];
}


int main(int argc, char *argv[]){
		float length_multiplier = 0.5; // (multiply the current line's length by this number
		int rotation_angle_degrees = 90; // (The amount to rotate per iteration) must be between 0 and 180
		int iterations = 5; // (number of iterations) Must be between 1 and 26 (both included) otherwise it uses too much memory, as memory usage is 10x2^iterations bytes

		const float line_length = 1;

		const unsigned long no_of_points = (1 << (iterations));

		// Declare sin and cosine maps
		/*float *sin_map, *cos_map;
		const int map_size = (360 + 360%32);
		const int map_physical_size = map_size * sizeof(float);
		cudaMalloc((void**) &sin_map, map_physical_size);
		cudaMalloc((void**) &cos_map, map_physical_size);
		*/
		// Declare angles and pointsX and pointsY lists
		short *angles;
		float *pointsX, *pointsY;
		const unsigned long short_list_size = no_of_points * sizeof(short);
		const unsigned long float_list_size = no_of_points * sizeof(float);

		float *minMax_X_Y; //Order: minX, minY, maxY (we do not need maxX as this is just minX*-1)
		float *minMaxWorkingList;
		
		hipMalloc((void**) &angles, short_list_size);
		hipMalloc((void**) &pointsX, float_list_size);
		hipMalloc((void**) &pointsY, float_list_size);

		hipMalloc((void**) &minMax_X_Y, sizeof(float) * 3);
		hipMalloc((void**) &minMaxWorkingList, float_list_size/2);
		
		// initilize the first 2 of angles, pointsX and pointsY
		short angles_host[] = {0, 0};
		float pointsX_host[] = {0, 0};
		float pointsY_host[] = {0, line_length};
		hipMemcpy(angles, angles_host, 2*sizeof(short), hipMemcpyHostToDevice);
		hipMemcpy(pointsX, pointsX_host, 2*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(pointsY, pointsY_host, 2*sizeof(float), hipMemcpyHostToDevice);

		//populate_sin_cos_maps<<<1, map_size>>>(sin_map, cos_map);

		unsigned int blocks = no_of_points/512 + (no_of_points % 512 != 0);
		cout << no_of_points << " " << blocks<<endl<<endl;
		calculate_points<<<1, 512>>>(angles, pointsX, pointsY, iterations-1, line_length/2, length_multiplier, rotation_angle_degrees);

		// make the host block until the device is finished with foo
  hipDeviceSynchronize();

		 // check for error
  hipError_t error = hipGetLastError();
  if(error != hipSuccess)
  {
    // print the CUDA error message and exit
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }else{
				cout << "ok for now" << endl;
		}

		
		//cudaFree(angles);

		calculateMin<<<1, 512>>>(pointsX, minMaxWorkingList, &minMax_X_Y[0], iterations, no_of_points/2);
		calculateMin<<<1, 512>>>(pointsY, minMaxWorkingList, &minMax_X_Y[1], iterations, no_of_points/2);
		calculateMax<<<1, 512>>>(pointsY, minMaxWorkingList, &minMax_X_Y[2], iterations, no_of_points/2);
		
		







		short *a = (short*)malloc(short_list_size);
		float *px = (float*)malloc(float_list_size);
		float *py = (float*)malloc(float_list_size);
		hipMemcpy(a, angles, short_list_size, hipMemcpyDeviceToHost);
		hipMemcpy(px, pointsX, float_list_size, hipMemcpyDeviceToHost);
		hipMemcpy(py, pointsY, float_list_size, hipMemcpyDeviceToHost);

		for(int i = 0; i < no_of_points; ++i){
				cout << i << " " << a[i] << " " << px[i] << " " << py[i] << endl;
		}


		cout << endl << endl;

		float *t = (float*)malloc(float_list_size/2);
		hipMemcpy(t, minMaxWorkingList, float_list_size/2, hipMemcpyDeviceToHost);

		for(int i = 0; i < no_of_points/2; ++i){
				cout << t[i] << endl;
		}

		float *minMax_X_Y_local = (float*)malloc(sizeof(float)*3);
		hipMemcpy(minMax_X_Y_local, minMax_X_Y, sizeof(float) * 3, hipMemcpyDeviceToHost);

		cout << endl << endl;
		for(int i = 0; i < 3; ++i){
				cout << minMax_X_Y_local[i] << endl;
		}
}
